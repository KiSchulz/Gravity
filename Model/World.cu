#include "hip/hip_runtime.h"
//
// Created by kis on 24.05.22.
//

#include <vector>
#include <chrono>

#include "World.cuh"

World::World(std::vector<Entity> entities) : size(entities.size()) {
    hipMalloc(&g_pos, sizeof(Vec3) * size);
    hipMalloc(&g_movDir, sizeof(Vec3) * size);
    hipMalloc(&g_mass, sizeof(float) * size);
    hipMalloc(&g_accVec, sizeof(Vec3) * size);

    posVec.resize(size);

    std::vector<Vec3> buffV3;
    buffV3.resize(size);
    for (std::size_t i = 0; i < size; i++) {
        buffV3[i] = entities[i].pos;
    }
    hipMemcpy(g_pos, buffV3.data(), sizeof(Vec3) * size, hipMemcpyHostToDevice);

    for (std::size_t i = 0; i < size; i++) {
        buffV3[i] = entities[i].velocity;
    }
    hipMemcpy(g_movDir, buffV3.data(), sizeof(Vec3) * size, hipMemcpyHostToDevice);

    std::vector<float> buff;
    buff.resize(size);
    for (std::size_t i = 0; i < size; i++) {
        buff[i] = entities[i].mass * gravityConstant;
    }
    hipMemcpy(g_mass, buff.data(), sizeof(float) * size, hipMemcpyHostToDevice);
}

World::~World() {
    hipFree(g_pos);
    hipFree(g_movDir);
    hipFree(g_mass);
    hipFree(g_accVec);
}

__global__ void calcAcc1(std::size_t size, const Vec3 *pos, const float *mass, Vec3 *accVec) {
    std::size_t index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        auto calcAcc = [pos, mass, index](std::size_t i) {
            Vec3 dist = pos[i] - pos[index];
            float len = dist.norm();

            return dist * (mass[i] / (len * len * len));
        };

        for (std::size_t i = 0; i < index; i++) {
            accVec[index] += calcAcc(i);
        }
        for (std::size_t i = index + 1; i < size; i++) {
            accVec[index] += calcAcc(i);
        }
    }
}

__global__ void updatePos(std::size_t size, const Vec3 *accVec, Vec3 *movDir, Vec3 *pos, float t) {
    std::size_t index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        movDir[index] += accVec[index] * t;
        pos[index] += movDir[index] * t;
    }
}

void World::step() {
    namespace sc = std::chrono;
    using namespace sc;
    time_point start = high_resolution_clock::now();

    std::size_t blockSize = 256;
    std::size_t numBlocks = (size + blockSize - 1) / blockSize;

    hipMemset(g_accVec, 0.0f, sizeof(Vec3) * size);
    calcAcc1<<<numBlocks, blockSize>>>(size, g_pos, g_mass, g_accVec);

    updatePos<<<numBlocks, blockSize>>>(size, g_accVec, g_movDir, g_pos, timeStepSize);

    timePassed += timeStepSize;

    hipDeviceSynchronize();

    speedup = timeStepSize / ((double) duration_cast<nanoseconds>(high_resolution_clock::now() - start).count() / 1e9f);
}

std::vector<Vec3> World::getPosVec() const {
    hipMemcpy((void *) posVec.data(), g_pos, sizeof(Vec3) * size, hipMemcpyDeviceToHost);
    return posVec;
}

double World::getSpeedup() const {
    return speedup;
}
