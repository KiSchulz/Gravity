#include "hip/hip_runtime.h"
//
// Created by kis on 24.05.22.
//

#include <vector>

#include "World.cuh"

World::World(std::vector<Entity> entities) : size(entities.size()) {
    hipMalloc(&g_pos, sizeof(float3) * size);
    hipMalloc(&g_movDir, sizeof(float3) * size);
    hipMalloc(&g_mass, sizeof(float) * size);

    std::vector<float3> buffV3;
    buffV3.resize(size);
    for (std::size_t i = 0; i < size; i++) {
        buffV3[i].x = entities[i].pos.x;
        buffV3[i].y = entities[i].pos.y;
        buffV3[i].z = entities[i].pos.z;
    }
    hipMemcpy(g_pos, buffV3.data(), sizeof(float3) * size, hipMemcpyHostToDevice);

    for (std::size_t i = 0; i < size; i++) {
        buffV3[i].x = entities[i].velocity.x;
        buffV3[i].y = entities[i].velocity.y;
        buffV3[i].z = entities[i].velocity.z;
    }
    hipMemcpy(g_movDir, buffV3.data(), sizeof(float3) * size, hipMemcpyHostToDevice);

    std::vector<float> buff;
    buff.resize(size);
    for (std::size_t i = 0; i < size; i++) {
        buff[i] = entities[i].mass;
    }
    hipMemcpy(g_mass, buff.data(), sizeof(float) * size, hipMemcpyHostToDevice);
}

World::~World() {
    hipFree(g_pos);
    hipFree(g_movDir);
    hipFree(g_mass);
}

std::vector<Entity> World::getEntities() const {
    std::vector<Entity> entities;
    std::vector<float3> buff;
    entities.resize(size);
    buff.resize(size);

    hipMemcpy(buff.data(), g_pos, sizeof(float3) * size, hipMemcpyDeviceToHost);

    for (std::size_t i = 0; i < size; i++) {
        entities[i].pos.x = buff[i].x;
        entities[i].pos.y = buff[i].y;
        entities[i].pos.z = buff[i].z;
    }

    return entities;
}

__device__ inline float3 operator+(const float3 &lhs, const float3 &rhs) {
    return {lhs.x + rhs.x, lhs.y + rhs.y, lhs.z + rhs.z};
}

__device__ inline float3 operator-(const float3 &lhs, const float3 &rhs) {
    return {lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z};
}

__device__ inline float3 operator*(const float3 &vec, float fac) {
    return {vec.x * fac, vec.y * fac, vec.z * fac};
}

__device__ inline float norm(const float3 &vec) {
    return sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__global__ void calcAcc(std::size_t size, const float3 *pos, const float *mass, float3 *accVec, float gravConst) {
    std::size_t index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        auto calcFG = [pos, mass, gravConst, index](std::size_t i) {
            float3 dist = pos[i] - pos[index];
            float len = norm(dist);

            float fg = (gravConst * mass[index] * mass[i]) / (len * len * len);

            return dist * fg;
        };

        accVec[index] = make_float3(0, 0, 0);

        for (std::size_t i = 0; i < index; i++) {
            accVec[index] = accVec[index] + calcFG(i);
        }
        for (std::size_t i = index + 1; i < size; i++) {
            accVec[index] = accVec[index] + calcFG(i);
        }

        accVec[index] = accVec[index] * (1 / mass[index]);
    }
}

__global__ void applyAcceleration(std::size_t size, const float3 *accVec, float3 *movDir, float t) {
    std::size_t index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        movDir[index] = movDir[index] + accVec[index] * t;
    }
}

__global__ void applyMovement(std::size_t size, const float3 *movDir, float3 *pos, float t) {
    std::size_t index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        pos[index] = pos[index] + movDir[index] * t;
    }
}

void World::step() {
    std::size_t blockSize = 256;
    std::size_t numBlocks = (size + blockSize - 1) / blockSize;
    float3 *accVec;
    hipMalloc(&accVec, size * sizeof(float3));

    calcAcc<<<numBlocks, blockSize>>>(size, g_pos, g_mass, accVec, gravityConstant);
    applyAcceleration<<<numBlocks, blockSize>>>(size, accVec, g_movDir, timeStepSize);

    hipFree(accVec);

    applyMovement<<<numBlocks, blockSize>>>(size, g_movDir, g_pos, timeStepSize);

    timePassed += timeStepSize;

    hipDeviceSynchronize();
}
