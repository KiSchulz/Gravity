#include "hip/hip_runtime.h"
//
// Created by kis on 24.05.22.
//

#include <vector>

#include "World.cuh"

World::World(std::vector<Entity> entities) : size(entities.size()) {
    hipMalloc(&g_pos, sizeof(Vec3) * size);
    hipMalloc(&g_movDir, sizeof(Vec3) * size);
    hipMalloc(&g_mass, sizeof(float) * size);

    std::vector<Vec3> buffV3;
    buffV3.resize(size);
    for (std::size_t i = 0; i < size; i++) {
        buffV3[i] = entities[i].pos;
    }
    hipMemcpy(g_pos, buffV3.data(), sizeof(Vec3) * size, hipMemcpyHostToDevice);

    for (std::size_t i = 0; i < size; i++) {
        buffV3[i] = entities[i].velocity;
    }
    hipMemcpy(g_movDir, buffV3.data(), sizeof(Vec3) * size, hipMemcpyHostToDevice);

    std::vector<float> buff;
    buff.resize(size);
    for (std::size_t i = 0; i < size; i++) {
        buff[i] = entities[i].mass;
    }
    hipMemcpy(g_mass, buff.data(), sizeof(float) * size, hipMemcpyHostToDevice);
}

World::~World() {
    hipFree(g_pos);
    hipFree(g_movDir);
    hipFree(g_mass);
}

std::vector<Entity> World::getEntities() const {
    std::vector<Entity> entities;
    std::vector<Vec3> buff;
    entities.resize(size);
    buff.resize(size);

    hipMemcpy(buff.data(), g_pos, sizeof(Vec3) * size, hipMemcpyDeviceToHost);

    for (std::size_t i = 0; i < size; i++) {
        entities[i].pos = buff[i];
    }

    return entities;
}

__device__ inline float3 operator+(const float3 &lhs, const float3 &rhs) {
    return {lhs.x + rhs.x, lhs.y + rhs.y, lhs.z + rhs.z};
}

__device__ inline float3 operator-(const float3 &lhs, const float3 &rhs) {
    return {lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z};
}

__device__ inline float3 operator*(const float3 &vec, float fac) {
    return {vec.x * fac, vec.y * fac, vec.z * fac};
}

__global__ void calcAcc(std::size_t size, const Vec3 *pos, const float *mass, Vec3 *accVec, float gravConst) {
    std::size_t index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        auto calcFG = [pos, mass, gravConst, index](std::size_t i) {
            Vec3 dist = pos[i] - pos[index];
            float len = norm3df(dist.x, dist.y, dist.z);

            float fg = (gravConst * mass[index] * mass[i]) / (len * len * len);

            return dist * fg;
        };

        accVec[index] = make_float3(0, 0, 0);

        for (std::size_t i = 0; i < index; i++) {
            accVec[index] = accVec[index] + calcFG(i);
        }
        for (std::size_t i = index + 1; i < size; i++) {
            accVec[index] = accVec[index] + calcFG(i);
        }

        accVec[index] = accVec[index] * (1 / mass[index]);
    }
}

__global__ void applyAcceleration(std::size_t size, const Vec3 *accVec, Vec3 *movDir, float t) {
    std::size_t index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        movDir[index] = movDir[index] + accVec[index] * t;
    }
}

__global__ void applyMovement(std::size_t size, const Vec3 *movDir, Vec3 *pos, float t) {
    std::size_t index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        pos[index] = pos[index] + movDir[index] * t;
    }
}

void World::step() {
    std::size_t blockSize = 256;
    std::size_t numBlocks = (size + blockSize - 1) / blockSize;
    float3 *accVec;
    hipMalloc(&accVec, size * sizeof(float3));

    calcAcc<<<numBlocks, blockSize>>>(size, g_pos, g_mass, accVec, gravityConstant);
    applyAcceleration<<<numBlocks, blockSize>>>(size, accVec, g_movDir, timeStepSize);

    hipFree(accVec);

    applyMovement<<<numBlocks, blockSize>>>(size, g_movDir, g_pos, timeStepSize);

    timePassed += timeStepSize;

    hipDeviceSynchronize();
}
